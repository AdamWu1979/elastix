#include "hip/hip_runtime.h"
/* $Id$ */
#include "cudaResampleImageFilter.cuh"
#include "CI/cubicPreFilter3D.cu"
#include "cudaInlineFunctions.h"

__constant__ float3 CUInputImageSpacing;
__constant__ float3 CUInputImageOrigin;
__constant__ float3 CUOutputImageSpacing;
__constant__ float3 CUOutputImageOrigin;
__constant__ float3 CUGridSpacing;
__constant__ float3 CUGridOrigin;
__constant__ int3   CUGridSize;
__constant__ float  CUDefaultPixelValue;

/* template linker errors... http://www.parashift.com/c++-faq-lite/templates.html#faq-35.14 */
template class cuda::CUDAResampleImageFilter<double, short, float>;
template class cuda::CUDAResampleImageFilter<double, int  , float>;
template class cuda::CUDAResampleImageFilter<double, float, float>;

#include "cudaDeformationsKernel.cu"

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
	::CUDAResampleImageFilter()
	: m_coeffsX(NULL)
	, m_coeffsY(NULL)
	, m_coeffsZ(NULL)
	, m_InputImage (NULL)
	, m_InputImageSize(make_int3(0,0,0))
	, m_Device(0)
	, m_MaxnrOfVoxelsPerIteration(1 << 20)
{
}

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
	::~CUDAResampleImageFilter()
{
}

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
void
	cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
	::cudaInit()
{
	checkExecutionParameters();
	cuda::hipSetDevice(m_Device);
	m_channelDescCoeff = hipCreateChannelDesc<TInternalImageType>();
}

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
void
cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
::cudaUnInit()
{
	cuda::hipUnbindTexture(m_tex_coeffsX);
	cuda::hipUnbindTexture(m_tex_coeffsY);
	cuda::hipUnbindTexture(m_tex_coeffsZ);
	cuda::hipUnbindTexture(m_tex_inputImage);
	cuda::hipFreeArray(m_coeffsX);
	cuda::hipFreeArray(m_coeffsY);
	cuda::hipFreeArray(m_coeffsZ);
	cuda::hipFreeArray(m_InputImage);
}

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
int
	cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
	::checkExecutionParameters()
{
	int deviceCount = 0;
	hipError_t err = hipGetDeviceCount(&deviceCount);
	return (err == hipSuccess) ? (deviceCount == 0) : 1;
}

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
void
	cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
	::cudaCopyImageSymbols(float3& InputImageSpacing, float3& InputImageOrigin, float3& OutputImageSpacing, float3& OutputImageOrigin, float DefaultPixelValue)
{
	/* copy some constant parameters to the GPU's constant cache */
	cuda::hipMemcpyToSymbol(HIP_SYMBOL(CUInputImageSpacing), InputImageSpacing,   hipMemcpyHostToDevice);
	cuda::hipMemcpyToSymbol(HIP_SYMBOL(CUInputImageOrigin),  InputImageOrigin,    hipMemcpyHostToDevice);

	cuda::hipMemcpyToSymbol(HIP_SYMBOL(CUOutputImageSpacing), OutputImageSpacing, hipMemcpyHostToDevice);
	cuda::hipMemcpyToSymbol(HIP_SYMBOL(CUOutputImageOrigin),  OutputImageOrigin,  hipMemcpyHostToDevice);

	cuda::hipMemcpyToSymbol(HIP_SYMBOL(CUDefaultPixelValue),  DefaultPixelValue,  hipMemcpyHostToDevice);
}

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
void
cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
::cudaCopyGridSymbols(float3& GridSpacing, float3& GridOrigin, int3& GridSize)
{
	/* copy some constant parameters to the GPU's constant cache */
	cuda::hipMemcpyToSymbol(HIP_SYMBOL(CUGridSpacing), GridSpacing, hipMemcpyHostToDevice);
	cuda::hipMemcpyToSymbol(HIP_SYMBOL(CUGridOrigin),  GridOrigin,  hipMemcpyHostToDevice);
	cuda::hipMemcpyToSymbol(HIP_SYMBOL(CUGridSize),    GridSize,    hipMemcpyHostToDevice);
}

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
void
	cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
	::cudaMallocTransformationData(int3 gridSize, const TInterpolatorPrecisionType* params)
{
	const int nrOfParametersPerDimension = gridSize.x * gridSize.y * gridSize.z;

	hipExtent gridExtent = make_hipExtent(gridSize.x, gridSize.y, gridSize.z);
	/* allocate memory on the GPU for the interpolation texture */
	cuda::hipMalloc3DArray(&m_coeffsX, &m_channelDescCoeff, gridExtent);
	cuda::hipMalloc3DArray(&m_coeffsY, &m_channelDescCoeff, gridExtent);
	cuda::hipMalloc3DArray(&m_coeffsZ, &m_channelDescCoeff, gridExtent);

	/* convert TInterpolatorPrecisionType to float, only thing textures support */
#if 1
	clock_t start = clock();
	TInternalImageType* params_tmp = new TInternalImageType[nrOfParametersPerDimension * 3];
	for (size_t i = 0; i != nrOfParametersPerDimension * 3; ++i) params_tmp[i] = static_cast<TInternalImageType>(params[i]);
	std::cout << "parameter type conversion took " << clock() - start << "ms" << std::endl;
	hipBindTextureToArray(m_coeffsX, &params_tmp[0 * nrOfParametersPerDimension], gridExtent, m_tex_coeffsX, m_channelDescCoeff);
	hipBindTextureToArray(m_coeffsY, &params_tmp[1 * nrOfParametersPerDimension], gridExtent, m_tex_coeffsY, m_channelDescCoeff);
	hipBindTextureToArray(m_coeffsZ, &params_tmp[2 * nrOfParametersPerDimension], gridExtent, m_tex_coeffsZ, m_channelDescCoeff);
	delete[] params_tmp;
#else
	/* there are some problems with Device2Device copy when src is not a pitched or 3D array... */
	TInternalImageType* params_gpu = cuda::hipMalloc<TInternalImageType>(nrOfParametersPerDimension);

	/* create the b-spline coefficients texture */
	cudaCastToType<TInterpolatorPrecisionType, TInternalImageType>(gridExtent, &params[0 * nrOfParametersPerDimension], params_gpu, hipMemcpyHostToDevice, m_Device);
	hipBindTextureToArray(m_coeffsX, params_gpu, gridExtent, m_tex_coeffsX, m_channelDescCoeff, false, true);
	cudaCastToType<TInterpolatorPrecisionType, TInternalImageType>(gridExtent, &params[1 * nrOfParametersPerDimension], params_gpu, hipMemcpyHostToDevice, m_Device);
	hipBindTextureToArray(m_coeffsY, params_gpu, gridExtent, m_tex_coeffsY, m_channelDescCoeff, false, true);
	cudaCastToType<TInterpolatorPrecisionType, TInternalImageType>(gridExtent, &params[2 * nrOfParametersPerDimension], params_gpu, hipMemcpyHostToDevice, m_Device);
	hipBindTextureToArray(m_coeffsZ, params_gpu, gridExtent, m_tex_coeffsZ, m_channelDescCoeff, false, true);
	cuda::hipFree(params_gpu);
#endif
}

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
void
	cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
	::cudaMallocImageData(int3 inputsize, int3 outputsize, const TImageType* data, bool PreFilter)
{
	m_InputImageSize            = inputsize;
	m_OutputImageSize           = outputsize;
	m_nrOfInputVoxels           = m_InputImageSize.x  * m_InputImageSize.y  * m_InputImageSize.z;
	m_MaxnrOfVoxelsPerIteration = min((size_t)(m_OutputImageSize.x * m_OutputImageSize.y * m_OutputImageSize.z), m_MaxnrOfVoxelsPerIteration);

	hipExtent volumeExtent     = make_hipExtent(m_InputImageSize.x, m_InputImageSize.y, m_InputImageSize.z);

	/* allocate in memory and PreFilter if requested. We do need to cast to float if not already 
	 * because linear filtering only works with floating point values */
	TInternalImageType* inputImage = cuda::hipMalloc<TInternalImageType>(m_nrOfInputVoxels);
	cudaCastToDevice(m_InputImageSize, data, inputImage);
	if (PreFilter)
	{
		CubicBSplinePrefilter3D(inputImage, volumeExtent.width, volumeExtent.height, volumeExtent.depth);
	}

	/* XXX - hipMemcpy3D fails if a DeviceToDevice copy src is not allocated with hipMallocPitch
	 * or hipMalloc3D, so we need this hack to get the data there */
	TInternalImageType* tmpImage = new TInternalImageType[m_nrOfInputVoxels];
	cuda::hipMemcpy(tmpImage, inputImage, m_nrOfInputVoxels, hipMemcpyDeviceToHost);
	cuda::hipFree(inputImage);

	/* create the image interpolation texture */
	cuda::hipMalloc3DArray(&m_InputImage, &m_channelDescCoeff, volumeExtent);
	hipBindTextureToArray(m_InputImage, tmpImage, volumeExtent, m_tex_inputImage, m_channelDescCoeff, false);
	delete[] tmpImage;

	/* allocate destination array */
	m_OutputImage = cuda::hipMalloc<TInternalImageType>(m_MaxnrOfVoxelsPerIteration);
}

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
void
	cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
	::GenerateData(TImageType* dst)
{
	/* split up applying the transformation due to memory constraints and make sure we never overflow the output image dimensions */
	const size_t nrOfOutputVoxels = m_OutputImageSize.x * m_OutputImageSize.y * m_OutputImageSize.z;
	dim3 dimBlock(256);
	dim3 dimGrid(m_MaxnrOfVoxelsPerIteration / dimBlock.x);
	size_t offset = 0;

	TInternalImageType* tmp_src = new TInternalImageType[m_MaxnrOfVoxelsPerIteration];
	if (nrOfOutputVoxels > m_MaxnrOfVoxelsPerIteration)
	{
		/* do a full run of m_MaxnrOfVoxelsPerIteration voxels */
		for (offset = 0; offset <= nrOfOutputVoxels - m_MaxnrOfVoxelsPerIteration; offset += m_MaxnrOfVoxelsPerIteration)
		{
			resample_image<<<dimGrid, dimBlock>>>(m_OutputImage, m_InputImageSize, m_OutputImageSize, offset);
			cuda::cudaCheckMsg("kernel launch failed: resample_image");
			cudaCastToHost(m_MaxnrOfVoxelsPerIteration, m_OutputImage, tmp_src, &dst[offset]);
		}
	}

	/* do the remainder ensuring again dimGrid*dimBlock is less than image size */
	dimGrid = dim3(nrOfOutputVoxels - offset) / dimBlock;
	resample_image<<<dimGrid, dimBlock>>>(m_OutputImage, m_InputImageSize, m_OutputImageSize, offset);
	cuda::cudaCheckMsg("kernel launch failed: resample_image");
	cudaCastToHost(dimGrid.x * dimBlock.x, m_OutputImage, tmp_src, &dst[offset]);

	/* do the final amount of voxels < dimBlock */
	offset += dimGrid.x * dimBlock.x;
	dimBlock = dim3(nrOfOutputVoxels - offset);
	dimGrid  = dim3(1);
	resample_image<<<dimGrid, dimBlock>>>(m_OutputImage, m_InputImageSize, m_OutputImageSize, offset);
	cuda::cudaCheckMsg("kernel launch failed: resample_image");
	cudaCastToHost(dimGrid.x * dimBlock.x, m_OutputImage, tmp_src, &dst[offset]);
	delete[] tmp_src;
}

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
template <typename TTextureType>
hipError_t
	cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
	::hipBindTextureToArray(hipArray* dst, const TInternalImageType* src, hipExtent& extent, TTextureType& tex, hipChannelFormatDesc& desc, bool normalized, bool onDevice)
{
	hipMemcpy3DParms copyParams = {0};
	copyParams.extent   = extent;
	copyParams.kind     = onDevice ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice;
	copyParams.dstArray = dst;
	copyParams.srcPtr   = make_hipPitchedPtr(const_cast<TInternalImageType*>(src), extent.width * sizeof(TInternalImageType), extent.width, extent.height);
	cuda::hipMemcpy3D(&copyParams);

	tex.normalized     = normalized;
	tex.filterMode     = cudaFilterMode;
	tex.addressMode[0] = tex.normalized ? hipAddressModeMirror: hipAddressModeClamp;
	tex.addressMode[1] = tex.normalized ? hipAddressModeMirror: hipAddressModeClamp;
	tex.addressMode[2] = tex.normalized ? hipAddressModeMirror: hipAddressModeClamp;
	return cuda::hipBindTextureToArray(tex, dst, desc);
}

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
void
cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
::cudaCastToHost(size_t size, const TInternalImageType* src, TInternalImageType* tmp_src, TImageType* dst)
{
	cuda::hipMemcpy(tmp_src, src, size, hipMemcpyDeviceToHost);
	for (size_t i = 0; i != size; ++i) dst[i] = static_cast<TImageType>(tmp_src[i]);
}

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
void
	cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
	::cudaCastToHost(int3 size, const TInternalImageType* src, TImageType* dst)
{
	hipExtent volumeExtent = make_hipExtent(size.x, size.y, size.z);
	cudaCastToType<TInternalImageType, TImageType>(volumeExtent, src, dst, hipMemcpyDeviceToHost, m_Device);
}

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
void
	cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
	::cudaCastToDevice(int3 size, const TImageType* src, TInternalImageType* dst)
{
	hipExtent volumeExtent = make_hipExtent(size.x, size.y, size.z);
	cudaCastToType<TImageType, TInternalImageType>(volumeExtent, src, dst, hipMemcpyHostToDevice, m_Device);
}

/* check for double TInputImageType or TOutputImageType */
template <class T> inline bool is_double();
template <class T> inline bool is_double() {return false;}
template <       > inline bool is_double<double>() {return true;}

template <>
float* cuda::cudaCastToType<float, float>(hipExtent& volumeExtent, const float* src, float* dst, hipMemcpyKind direction, int device)
{
	const size_t voxelsPerSlice = volumeExtent.width * volumeExtent.height;
	cuda::hipMemcpy(dst, src, voxelsPerSlice * volumeExtent.depth, direction);
	return dst;
}

template <class TInputImageType, class TOutputImageType>
TOutputImageType* cuda::cudaCastToType(hipExtent& volumeExtent, const TInputImageType* src, TOutputImageType* dst, hipMemcpyKind direction, int device)
{
	const size_t voxelsPerSlice = volumeExtent.width * volumeExtent.height;
	size_t offset = 0;
	dim3 dimBlock(min((int)max(volumeExtent.width, volumeExtent.height), 512));
	dim3 dimGrid(voxelsPerSlice / dimBlock.x);
	/* not a perfect fit, fix it */
	if (dimBlock.x * dimGrid.x != voxelsPerSlice) ++dimGrid.x;

	clock_t start = clock();
	if (is_double<TInputImageType>() || is_double<TOutputImageType>())
	{
		hipDeviceProp_t prop;
		cuda::hipGetDeviceProperties(&prop, device);
		/* only devices from compute capability 1.3 support double precision on the device */
		if (prop.major == 1 && prop.minor < 3)
		{
			const size_t nof_elements  = volumeExtent.width * volumeExtent.height * volumeExtent.depth;
			switch (direction)
			{
			case hipMemcpyHostToDevice: {
				if (is_double<TOutputImageType>()) throw itk::ExceptionObject("GPU doesn't support double-precision");

				/* we can still convert from double (TInputImageType) to TOutputImageType, just not on the GPU */
				TOutputImageType* src_cast = new TOutputImageType[nof_elements];
				for (size_t i = 0; i != nof_elements; ++i) src_cast[i] = static_cast<TOutputImageType>(src[i]);
				cuda::hipMemcpy(dst, src_cast, nof_elements, hipMemcpyHostToDevice);
			} break;
			case hipMemcpyDeviceToHost: {
				if (is_double<TInputImageType>()) throw itk::ExceptionObject("GPU doesn't support double-precision");

				/* we can still convert from TOutputImageType to double (TInputImageType), just not on the GPU */
				TInputImageType* dst_cast = new TInputImageType[nof_elements];
				cuda::hipMemcpy(dst_cast, src, nof_elements, hipMemcpyDeviceToHost);
				for (size_t i = 0; i != nof_elements; ++i) dst[i] = static_cast<TOutputImageType>(dst_cast[i]);
			} break;
			}

			goto END_OF_FUNCTION;
		}
	}

	switch (direction)
	{
	case hipMemcpyHostToDevice: {
		TInputImageType* tmp = cuda::hipMalloc<TInputImageType>(voxelsPerSlice);
		for (int slice = 0; slice != volumeExtent.depth; ++slice, offset += voxelsPerSlice)
		{
			cuda::hipMemcpy(tmp, src + offset, voxelsPerSlice, hipMemcpyHostToDevice);
			cast_to_type<TInputImageType, TOutputImageType><<<dimGrid, dimBlock>>>(dst + offset, tmp, voxelsPerSlice);
			cuda::cudaCheckMsg("kernel launch failed: cast_to_type");
		}
		hipFree(tmp);
		} break;
	case hipMemcpyDeviceToHost: {
		TOutputImageType* tmp = cuda::hipMalloc<TOutputImageType>(voxelsPerSlice);
		for (int slice = 0; slice != volumeExtent.depth; ++slice, offset += voxelsPerSlice)
		{
			cast_to_type<TInputImageType, TOutputImageType><<<dimGrid, dimBlock>>>(tmp, src + offset, voxelsPerSlice);
			cuda::cudaCheckMsg("kernel launch failed: cast_to_type");
			cuda::hipMemcpy(dst + offset, tmp, voxelsPerSlice, hipMemcpyDeviceToHost);
		}
		hipFree(tmp);
		} break;
	}

END_OF_FUNCTION:
	std::cout << "type conversion took " << clock() - start << "ms" << std::endl;
	return dst;
}
